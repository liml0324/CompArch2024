#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

#ifndef BLOCK_LEN
#define BLOCK_LEN 32
#endif
// #define VERIFY

#ifndef N
#define N (1 << 14)
#endif

__global__ void gemm_baseline(float *A, float *B, float *C, int n);
void gemm_verify(float *A, float *B, float *C);

using namespace std;

int main()
{
    // malloc A, B, C
    float *A = (float *)malloc(N * N * sizeof(float));
    float *B = (float *)malloc(N * N * sizeof(float));
    float *C = (float *)malloc(N * N * sizeof(float));
    // random initialize A, B
    default_random_engine generator((unsigned)time(NULL));
    uniform_real_distribution<float> distribution(-1, 1);
    for (int i = 0; i < N * N; i++) {
        A[i] = distribution(generator);
        B[i] = distribution(generator);
    }
    // cumalloc A, B, C
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N * N * sizeof(float));
    hipMalloc((void**)&d_B, N * N * sizeof(float));
    hipMalloc((void**)&d_C, N * N * sizeof(float));
    // define gridsize and blocksize
    dim3 gridsize(N/BLOCK_LEN, N/BLOCK_LEN);
    dim3 blocksize(BLOCK_LEN, BLOCK_LEN);
    // copy A, B to device
    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);
    // launch kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    gemm_baseline<<<gridsize, blocksize>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float *D = (float *)malloc(N * N * sizeof(float));
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time: " << milliseconds << "ms" << endl;
    hipMemcpy(D, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);
    // compute
    #ifdef VERIFY
    gemm_verify(A, B, C);
    for (int i = 0; i < N * N; i++) {
        if (abs(C[i] - D[i]) > 1e-3) {
            cout << "Error: " << i << " " << C[i] << " " << D[i] << endl;
            break;
        }
    }
    free(D);
    #endif
    // free mem
    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
__global__ void gemm_baseline(float* A, float * B, float* C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row >= n || col >= n) {
        return;
    }
    float sum = 0;
    for (int k = 0; k < n; k++) {
        sum += A[row * n + k] * B[k * n + col];
    }
    C[row * n + col] = sum;
}
void gemm_verify(float *A, float *B, float *C) {
    memset(C, 0, N * N * sizeof(float));
    for (int i = 0; i < N; i++) {
        for (int k = 0; k < N; k++) {
            for (int j = 0; j < N; j++) {
                C[i * N + j] += A[i * N + k] * B[k * N + j];
            }
        }
    }
}
