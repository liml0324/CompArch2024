#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

#define BLOCK_LEN 32
// #define VERIFY

#define N (1 << 15)

__global__ void gemm_block(float *A, float *B, float *C, int n);
void gemm_verify(float *A, float *B, float *C);

using namespace std;

int main() {
    // malloc A, B, C
    float *A = (float *)malloc(N * N * sizeof(float));
    float *B = (float *)malloc(N * N * sizeof(float));
    float *C = (float *)malloc(N * N * sizeof(float));
    // random initialize A, B
    default_random_engine generator((unsigned)time(NULL));
    uniform_real_distribution<float> distribution(-1, 1);
    for (int i = 0; i < N * N; i++) {
        A[i] = distribution(generator);
        B[i] = distribution(generator);
    }
    // cumalloc A, B, C
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N * N * sizeof(float));
    hipMalloc((void**)&d_B, N * N * sizeof(float));
    hipMalloc((void**)&d_C, N * N * sizeof(float));
    // define gridsize and blocksize
    dim3 gridsize(N/BLOCK_LEN, N/BLOCK_LEN);
    dim3 blocksize(BLOCK_LEN, BLOCK_LEN);
    // copy A, B to device
    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);
    // launch kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    gemm_block<<<gridsize, blocksize>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float *D = (float *)malloc(N * N * sizeof(float));
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time: " << milliseconds << "ms" << endl;
    hipMemcpy(D, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);
    // compute
    #ifdef VERIFY
    gemm_verify(A, B, C);
    for (int i = 0; i < N * N; i++) {
        if (abs(C[i] - D[i]) > 1e-3) {
            cout << "Error: " << i << " " << C[i] << " " << D[i] << endl;
            break;
        }
    }
    free(D);
    #endif
    // free mem
    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}

__global__ void gemm_block(float *A, float *B, float *C, int n) {
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * BLOCK_LEN + ty;
    int col = bx * BLOCK_LEN + tx;

    __shared__ float As[BLOCK_LEN][BLOCK_LEN];
    __shared__ float Bs[BLOCK_LEN][BLOCK_LEN];

    float Csub = 0;

    for (int t = 0; t < n / BLOCK_LEN; t++) {
        As[ty][tx] = A[row * n + t * BLOCK_LEN + tx];
        Bs[ty][tx] = B[(t * BLOCK_LEN + ty) * n + col];
        __syncthreads();

        for (int i = 0; i < BLOCK_LEN; i++) {
            Csub += As[ty][i] * Bs[i][tx];
        }
        __syncthreads();
    }

    C[row * n + col] = Csub;

}

void gemm_verify(float *A, float *B, float *C) {
    memset(C, 0, N * N * sizeof(float));
    for (int i = 0; i < N; i++) {
        for (int k = 0; k < N; k++) {
            for (int j = 0; j < N; j++) {
                C[i * N + j] += A[i * N + k] * B[k * N + j];
            }
        }
    }
}